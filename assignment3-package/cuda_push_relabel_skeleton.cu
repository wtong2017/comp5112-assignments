#include "hip/hip_runtime.h"
/**
 * Name: Tong Wai
 * Student id: 20271356
 * ITSC email: wtong@connect.ust.hk
 */

#include <cstring>
#include <cstdint>
#include <cstdlib>

#include <vector>
#include <iostream>

#include "cuda_push_relabel.h"

using namespace std;

/*
 *  You can add helper functions and variables as you wish.
 */
void pre_flow(int *dist, int64_t *excess, int *cap, int *flow, int N, int src) {
    dist[src] = N;
    for (auto v = 0; v < N; v++) {
        flow[utils::idx(src, v, N)] = cap[utils::idx(src, v, N)];
        flow[utils::idx(v, src, N)] = -flow[utils::idx(src, v, N)];
        excess[v] = flow[utils::idx(src, v, N)];
    }
}

__global__ void push(int *active_nodes, int *cap, int *flow, int *dist, int64_t *excess, int64_t *stash_excess, int active_nodes_size, int N) {
    int num_threads = gridDim.x * blockDim.x;
    int my_rank = blockDim.x * blockIdx.x + threadIdx.x;

    int avg = (active_nodes_size + num_threads - 1) / num_threads;
    int nodes_beg = avg * my_rank;
    int nodes_end = min(avg * (my_rank + 1), active_nodes_size);

    // printf("Hello from block %d, thread %d, begin %d, end %d \n", blockIdx.x, threadIdx.x, nodes_beg, nodes_end);
    for (auto nodes_it = nodes_beg; nodes_it < nodes_end; nodes_it++) {
        auto u = active_nodes[nodes_it];
        for (auto v = 0; v < N; v++) {
            auto residual_cap = cap[utils::dev_idx(u, v, N)] -
                                flow[utils::dev_idx(u, v, N)];
            if (residual_cap > 0 && dist[u] > dist[v] && excess[u] > 0) {
                auto send = (excess[u] - residual_cap > 0 ? residual_cap : excess[u]);
                flow[utils::dev_idx(u, v, N)] += send;
                flow[utils::dev_idx(v, u, N)] -= send;
                excess[u] -= send;
                stash_excess[v] += send;
            }
        }
    }

    // for (int i = 0; i < N; i++) {
    //     printf("%d, ", excess[i]);
    // }
    // printf("\n");

    // for (int i = 0; i < N; i++) {
    //     cout << stash_excess[i] << ", ";
    // }
    // cout << endl;

    // for (int i = 0; i < N; i++) {
    //     for (int j = 0; j < N; j++) {
    //         cout << flow[utils::idx(i, j, N)] << ", ";
    //     }
    //     cout << endl;
    // }

}

int push_relabel(int blocks_per_grid, int threads_per_block, int N, int src, int sink, int *cap, int *flow) {
    /*
     *  Please fill in your codes here.
     */
    int *dist = (int *) calloc(N, sizeof(int));
    int *stash_dist = (int *) calloc(N, sizeof(int));
    auto *excess = (int64_t *) calloc(N, sizeof(int64_t));
    auto *stash_excess = (int64_t *) calloc(N, sizeof(int64_t));

    size_t sizeNNInt = N * N * sizeof(int);
    size_t sizeNInt = N * sizeof(int);
    size_t sizeNInt64 = N * sizeof(int64_t);
    int *d_cap, *d_flow, *d_dist;
    int64_t *d_excess, *d_stash_excess;

    // PreFlow
    pre_flow(dist, excess, cap, flow, N, src);

    hipMalloc(&d_cap, sizeNNInt);
    hipMalloc(&d_flow, sizeNNInt);
    hipMalloc(&d_dist, sizeNInt);
    // hipMalloc(&d_stash_dist, sizeNInt);
    hipMalloc(&d_excess, sizeNInt64);
    hipMalloc(&d_stash_excess, sizeNInt64);

    hipMemcpy(d_cap, cap, sizeNNInt, hipMemcpyHostToDevice);
    hipMemcpy(d_flow, flow, sizeNNInt, hipMemcpyHostToDevice);

    vector<int> active_nodes;
    int *stash_send = (int *) calloc(N * N, sizeof(int));
    int *d_active_nodes;
    for (auto u = 0; u < N; u++) {
        if (u != src && u != sink) {
            active_nodes.emplace_back(u);
        }
    }

    int counter = 0;
    // Four-Stage Pulses.
    while (!active_nodes.empty()) {
        // if (counter > 3)
        //     break;
        int active_nodes_size = active_nodes.size();
        hipMalloc(&d_active_nodes, sizeof(int) * active_nodes_size);
        hipMemcpy(d_active_nodes, &active_nodes[0], sizeof(int) * active_nodes_size, hipMemcpyHostToDevice);

        hipMemcpy(d_dist, dist, sizeNInt, hipMemcpyHostToDevice);
        hipMemcpy(d_excess, excess, sizeNInt64, hipMemcpyHostToDevice);
        hipMemcpy(d_stash_excess, stash_excess, sizeNInt64, hipMemcpyHostToDevice);

        // Stage 1: push.
        // for (auto u : active_nodes) {
        //     for (auto v = 0; v < N; v++) {
        //         auto residual_cap = cap[utils::idx(u, v, N)] -
        //                             flow[utils::idx(u, v, N)];
        //         if (residual_cap > 0 && dist[u] > dist[v] && excess[u] > 0) {
        //             stash_send[utils::idx(u, v, N)] = std::min<int64_t>(excess[u], residual_cap);
        //             excess[u] -= stash_send[utils::idx(u, v, N)];
        //         }
        //     }
        // }
        // for (auto u : active_nodes) {
        //     for (auto v = 0; v < N; v++) {
        //         if (stash_send[utils::idx(u, v, N)] > 0) {
        //             flow[utils::idx(u, v, N)] += stash_send[utils::idx(u, v, N)];
        //             flow[utils::idx(v, u, N)] -= stash_send[utils::idx(u, v, N)];
        //             stash_excess[v] += stash_send[utils::idx(u, v, N)];
        //             stash_send[utils::idx(u, v, N)] = 0;
        //         }
        //     }
        // }

        push<<<blocks_per_grid, threads_per_block>>>(d_active_nodes, d_cap, d_flow, d_dist, d_excess, d_stash_excess, active_nodes_size, N);

        hipMemcpy(flow, d_flow, sizeNNInt, hipMemcpyDeviceToHost);
        hipMemcpy(excess, d_excess, sizeNInt64, hipMemcpyDeviceToHost);
        hipMemcpy(stash_excess, d_stash_excess, sizeNInt64, hipMemcpyDeviceToHost);

        // if (counter == 2) {
        //     for (int i = 0; i < N; i++) {
        //         cout << excess[i] << ", ";
        //     }
        //     cout << endl;
        // }

        // for (int i = 0; i < N; i++) {
        //     cout << stash_excess[i] << ", ";
        // }
        // cout << endl;

        // for (int i = 0; i < N; i++) {
        //     for (int j = 0; j < N; j++) {
        //         cout << flow[utils::idx(i, j, N)] << ", ";
        //     }
        //     cout << endl;
        // }
        // break;

        // Stage 2: relabel (update dist to stash_dist).
        memcpy(stash_dist, dist, N * sizeof(int));
        for (auto u : active_nodes) {
            if (excess[u] > 0) {
                int min_dist = INT32_MAX;
                for (auto v = 0; v < N; v++) {
                    auto residual_cap = cap[utils::idx(u, v, N)] - flow[utils::idx(u, v, N)];
                    if (residual_cap > 0) {
                        min_dist = min(min_dist, dist[v]);
                        stash_dist[u] = min_dist + 1;
                    }
                }
            }
        }

        // Stage 3: update dist.
        swap(dist, stash_dist);

        // Stage 4: apply excess-flow changes for destination vertices.
        for (auto v = 0; v < N; v++) {
            if (stash_excess[v] != 0) {
                excess[v] += stash_excess[v];
                stash_excess[v] = 0;
            }
        }

        // Construct active nodes.
        hipFree(d_active_nodes);
        active_nodes.clear();
        for (auto u = 0; u < N; u++) {
            if (excess[u] > 0 && u != src && u != sink) {
                active_nodes.emplace_back(u);
            }
        }
        counter++;
        // printf("Finish %d\n", counter);
    }

    hipFree(d_cap);
    hipFree(d_flow);
    hipFree(d_excess);

    free(dist);
    free(stash_dist);
    free(excess);
    free(stash_excess);
    free(stash_send);

    return 0;
}
